#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

/* CUBLAS code 
static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}
*/
/*
 * Host code
 */

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare most variables.*/
    mxGPUArray const *A, *B;
    mxGPUArray *C;
    const half *d_A, *d_B;
    const half **d_Aarr, **d_Barr;
    half *d_C;
    half const alpha = 1, beta = 0;
    hipStream_t *streamArray = 0;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    /* we expect 2 uint16 gpu Arrays*/

    if (nrhs!=2) {
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:WrongNumberOfInputs", "Expected 2 inputs. The 'transpose' and 'ctranspose' options are not supported yet.");
    } else if( !mxIsGPUArray(prhs[0]) || !mxIsGPUArray(prhs[1]) ) {
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:WrongInputType", "Expected arguments 1 and 2 to be gpuArray types.");
    }

    // get the actual matrix gpu reference
    A = mxGPUCreateFromMxArray(prhs[0]); // first matrix 
    B = mxGPUCreateFromMxArray(prhs[1]); // second matrix

    const mwSize nDimsA = mxGPUGetNumberOfDimensions(A);
    const mwSize * dimsA = mxGPUGetDimensions(A);
    const mwSize nDimsB = mxGPUGetNumberOfDimensions(B);
    const mwSize * dimsB = mxGPUGetDimensions(B);
    /* Throw an error if the input is not a GPU array. */
    if(nDimsA < 2){
        mxGPUDestroyGPUArray(A); // cleanup
        mxGPUDestroyGPUArray(B); // cleanup
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:EmptyInput", "Expected input 1 to have 2 or more dimensions.");
    } else if(nDimsB < 2){
        mxGPUDestroyGPUArray(A); // cleanup
        mxGPUDestroyGPUArray(B); // cleanup
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:EmptyInput", "Expected input 2 to have 2 or more dimensions.");
    } else if(dimsA[1]!= dimsB[0]){
        mxGPUDestroyGPUArray(A); // cleanup
        mxGPUDestroyGPUArray(B); // cleanup
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:MatrixMultiplicationDimensions", "The size of A in dimension 2 must match the size of B in dimension 1.");
    }

    // matrix array size
    const int M = dimsA[0], K = dimsA[1], N = dimsB[1];

    /*
     * Verify that A really is a uint16 array before extracting the pointer.
     % I think this is unnecessary?
     */
    if (mxGPUGetClassID(A) != mxUINT16_CLASS || mxGPUGetClassID(B) != mxUINT16_CLASS) {
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:WrongInputType", "Expected underlying type to be uint16.");
        mxGPUDestroyGPUArray(A); // cleanup
        mxGPUDestroyGPUArray(B); // cleanup
    } else if (mxGPUGetComplexity(A) != mxREAL) {
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:WrongComplexity", "Expected input 1 to be real.");
    } else if (mxGPUGetComplexity(B) != mxREAL) {
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:WrongComplexity", "Expected input 2 to be real.");        
    }
    d_A = (const half *)(mxGPUGetDataReadOnly(A));
    d_B = (const half *)(mxGPUGetDataReadOnly(B));
    
    
    // get the output matrix size
    const mwSize nDimsC = max(nDimsA, nDimsB); // number of output dimensions
    mwSize * dimsC = (mwSize *)mxMalloc(nDimsC * sizeof(nDimsC)); // array for each dimension size
    dimsC[0] = M;
    dimsC[1] = N;
    for(int d = 2; d < nDimsC; ++d) // for dims 3+
        dimsC[d] = max((d < nDimsA ? dimsA[d] : 1), (d < nDimsB ? dimsB[d] : 1)); // new size is max of either size

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    C = mxGPUCreateGPUArray(nDimsC,
                            dimsC,
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    const size_t Csz = (size_t) mxGPUGetNumberOfElements(C); // total size of the data
    d_C = (half *)(mxGPUGetData(C)); // point to device data
    
    // make sure that the number of output strides matches the size of the data that we computed
    const size_t L = Csz / M / N; // number of strides we need to find

    /* we need to generate a set of pointers that point to the 
       location of the data for each stride, 
       while broadcasting over dimensions */
    d_Aarr = (const half **) mxMalloc(L * sizeof(d_Aarr));
    for(int i = 0; i < L; ++i){
        size_t szA = 1, szC = 1; // size so far
        d_Aarr[i] = d_A; // initial pointer locations
        for(int d = 2; d < nDimsA; ++d){ // for each upper dim
            const size_t ind = (dimsA[d] == 1) ? 0 : ((i / szC) % dimsC[d]); // index for this dim
            d_Aarr[i] += (ind*M*K*szA); // increment pointer
            szA *= dimsA[d]; // increment stride size
            szC *= dimsC[d]; // increment stride size
        }
    }

    d_Barr = (const half **) mxMalloc(L * sizeof(d_Barr));
    for(int i = 0; i < L; ++i){
        size_t szB = 1, szC = 1; // size so far
        d_Barr[i] = d_B; // initial pointer locations
        for(int d = 2; d < nDimsB; ++d){ // for each upper dim
            const size_t ind = (dimsB[d] == 1) ? 0 : ((i / szC) % dimsC[d]); // index for this dim
            d_Barr[i] += (ind*K*N*szB); // increment pointer
            szB *= dimsB[d]; // increment stride size
            szC *= dimsC[d]; // increment stride size
        }
    }

    /* code to call CUBLAS */
    hipError_t cudaErr; // error
    hipblasStatus_t stat; // status
    hipblasHandle_t handle; // handle
    hipblasOperation_t trans = HIPBLAS_OP_N; // ONE OF CUBLAS_OP_{N,T,C} for none/trans/ctrans
    streamArray = (hipStream_t *) mxMalloc(L * sizeof(hipStream_t *)); // new stream for each call - we assume this is less than the max
    stat = hipblasCreate(&handle);
    
    // create a set of new streams
    int i_stream;
    for (i_stream = 0; i_stream < L; ++i_stream){
        cudaErr = hipStreamCreateWithFlags(&streamArray[i_stream], hipStreamNonBlocking);
        if(cudaErr != hipSuccess)
            break;
    }

    // launch a matrix multiply on each stream
    for(int i = 0; i < L; ++i) // for each output index
        if (stat == HIPBLAS_STATUS_SUCCESS && cudaErr == hipSuccess) { // we succeeded in the last call: 
            // hipblasSetStream(handle, streamArray[i]); // move to the next stream
            stat = hipblasHgemm(handle, trans, trans, M,N,K, &alpha, // call gemm for dense matrix x matrix multiply
                           d_Aarr[i], M, d_Barr[i], K, &beta, d_C + i*M*N, M);
        }

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(C);

    /* -------------- CLEANUP ------------ */
    // cleanup CUBLAS (only as many streams as were created!)
    for (int i = 0; i < i_stream; ++i){
        cudaErr = hipStreamDestroy(streamArray[i]);
    }
    mxFree(streamArray);
    hipblasDestroy(handle); 
    
    // Free temporary array allocations
    mxFree(d_Aarr);
    mxFree(d_Barr);
    mxFree(dimsC);
    
    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(A); // cleanup
    mxGPUDestroyGPUArray(B); // cleanup
    mxGPUDestroyGPUArray(C); // cleanup

    if(stat != HIPBLAS_STATUS_SUCCESS){
        mexErrMsgIdAndTxt("parallel:gpu:pagemtimes:failure", "Failed to call the CUDA kernels.");
    }
}
